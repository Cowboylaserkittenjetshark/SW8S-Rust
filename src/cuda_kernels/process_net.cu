
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdint.h>
#include <stdio.h>

#define MAX_THREADS (512)
#define WARP_SIZE (32)

struct CudaFormatMat {
  int32_t rows;
  int32_t cols;
  float *bytes;
};
struct YoloDetectionCuda {
  double confidence;
  double x;
  double y;
  double width;
  double height;
  int32_t class_id;
};

__global__ void process_net(uintptr_t block_count,
                            YoloDetectionCuda *processed_detects,
                            bool *processed_valid) {
  auto id = blockIdx.x * blockDim.x + threadIdx.x;

  // Get rid of leftover threads
  if (id >= block_count)
    return;

  processed_valid[id] = false;
}

extern "C" {
int process_net_kernel(CudaFormatMat *const result, uintptr_t const num_levels,
                       double const threshold,
                       YoloDetectionCuda *processed_detects,
                       bool *processed_valid, uintptr_t const total_rows) {

  hipStream_t kernel_stream;
  hipStreamCreate(&kernel_stream);

  YoloDetectionCuda *processed_detects_cuda;
  bool *processed_valid_cuda;
  hipMallocAsync(&processed_detects_cuda,
                  sizeof(YoloDetectionCuda) * total_rows, kernel_stream);
  hipMallocAsync(&processed_valid_cuda, sizeof(bool) * total_rows,
                  kernel_stream);

  uintptr_t row_offset = 0;
  for (uintptr_t i = 0; i < num_levels; ++i) {
    CudaFormatMat *mat = result + i;
    auto num_rows = mat->rows;
    auto mat_size = sizeof(num_rows * mat->cols);
    float *mat_bytes;

    hipMallocAsync(&mat_bytes, mat_size, kernel_stream);
    hipMemcpyAsync(&mat_bytes, &mat, mat_size, hipMemcpyHostToDevice,
                    kernel_stream);

    int32_t blocksize = MAX_THREADS;
    int32_t block_count;
    if (num_rows < blocksize) {
      blocksize = num_rows;
      block_count = 1;
    } else {
      // Ceiling divide, from https://stackoverflow.com/a/14878734
      block_count = num_rows / MAX_THREADS + (num_rows % MAX_THREADS != 0);
    }

    process_net<<<block_count, blocksize, 0, kernel_stream>>>(
        block_count, processed_detects_cuda + row_offset,
        processed_valid_cuda + row_offset);

    hipFreeAsync(mat_bytes, kernel_stream);

    row_offset += num_rows;
  }

  hipMemcpyAsync(processed_detects, processed_detects_cuda,
                  sizeof(YoloDetectionCuda) * total_rows,
                  hipMemcpyDeviceToHost, kernel_stream);
  hipMemcpyAsync(processed_valid, processed_valid_cuda,
                  sizeof(bool) * total_rows, hipMemcpyDeviceToHost,
                  kernel_stream);

  hipStreamSynchronize(kernel_stream);

  return 0;
}
}
