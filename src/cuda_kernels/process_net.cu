
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdio>
#include <stdio.h>

__global__ void process_net(int *result) {
  printf("Hello World from GPU!\n");
  *result = 1 + 1;
}

extern "C" {
int process_net_kernel() {

  hipStream_t kernel_stream;
  hipStreamCreate(&kernel_stream);

  int *result;
  hipMallocAsync(&result, sizeof(int), kernel_stream);

  process_net<<<1, 1, 0, kernel_stream>>>(result);

  int local_result;

  hipMemcpyAsync(&local_result, result, sizeof(int), hipMemcpyDeviceToHost,
                  kernel_stream);
  hipFreeAsync(result, kernel_stream);
  hipStreamSynchronize(kernel_stream);

  assert(local_result == 2);

  return 0;
}
}
