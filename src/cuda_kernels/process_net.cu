
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdint.h>
#include <stdio.h>

#define MAX_THREADS (512)
#define WARP_SIZE (32)

struct CudaFormatMat {
  int32_t rows;
  int32_t cols;
  float *bytes;
};
struct YoloDetectionCuda {
  double confidence;
  double x;
  double y;
  double width;
  double height;
  int32_t class_id;
};

__forceinline__ __device__ float adjust_base(uintptr_t idx, float const factor,
                                             float const *row_bytes) {
  return row_bytes[idx] * factor;
}

__forceinline__ __device__ float x_adjust(uintptr_t idx, float const factor,
                                          float const *row_bytes) {
  return (adjust_base(idx, factor, row_bytes) / 640.0) * 800.0;
}

__forceinline__ __device__ float y_adjust(uintptr_t idx, float const factor,
                                          float const *row_bytes) {
  return (adjust_base(idx, factor, row_bytes) / 640.0) * 600.0;
}

__global__ void process_net(uintptr_t num_rows, uintptr_t num_cols,
                            float const threshold, float const factor,
                            float const *mat_bytes,
                            YoloDetectionCuda *processed_detects,
                            bool *processed_valid) {
  auto id = blockIdx.x * blockDim.x + threadIdx.x;

  // Get rid of leftover threads
  if (id >= num_rows)
    return;

  float const *row = mat_bytes + (id * num_cols);

  float confidence = row[4];
  bool valid = confidence > threshold;
  processed_valid[id] = valid;

  // Skip remaining processing for invalid
  if (!valid)
    return;

  // Start at offset in data, then shift to starting at 0.
  uintptr_t class_id = 5;
  float class_value = row[class_id];
  for (uintptr_t i = 6; i < num_cols; ++i) {
    if (class_value < row[i]) {
      class_id = i;
      class_value = row[i];
    }
  }
  class_id -= 5;

  float center_x = x_adjust(0, factor, row);
  float center_y = y_adjust(1, factor, row);
  float width = x_adjust(2, factor, row);
  float height = y_adjust(3, factor, row);

  float left = center_x - (width / 2.0);
  float top = center_y - (height / 2.0);

  processed_detects[id] = YoloDetectionCuda{
      confidence, left, top, width, height, static_cast<int32_t>(class_id)};
}

extern "C" {
int process_net_kernel(CudaFormatMat *const result, uintptr_t const num_levels,
                       float const threshold, float const factor,
                       uintptr_t const total_rows,
                       YoloDetectionCuda *processed_detects,
                       bool *processed_valid) {

  hipStream_t kernel_stream;
  hipStreamCreate(&kernel_stream);

  YoloDetectionCuda *processed_detects_cuda;
  bool *processed_valid_cuda;
  hipMallocAsync(&processed_detects_cuda,
                  sizeof(YoloDetectionCuda) * total_rows, kernel_stream);
  hipMallocAsync(&processed_valid_cuda, sizeof(bool) * total_rows,
                  kernel_stream);

  uintptr_t row_offset = 0;
  for (uintptr_t i = 0; i < num_levels; ++i) {
    CudaFormatMat *mat = result + i;
    auto num_rows = mat->rows;
    uintptr_t num_cols = static_cast<uintptr_t>(mat->cols);
    auto mat_size = num_rows * num_cols * sizeof(float);
    float *mat_bytes;

    hipMallocAsync(&mat_bytes, mat_size, kernel_stream);
    hipMemcpyAsync(mat_bytes, mat->bytes, mat_size, hipMemcpyHostToDevice,
                    kernel_stream);

    int32_t blocksize = MAX_THREADS;
    int32_t block_count;
    if (num_rows < blocksize) {
      blocksize = num_rows;
      block_count = 1;
    } else {
      // Ceiling divide, from https://stackoverflow.com/a/14878734
      block_count = num_rows / MAX_THREADS + (num_rows % MAX_THREADS != 0);
    }

    process_net<<<block_count, blocksize, 0, kernel_stream>>>(
        num_rows, num_cols, threshold, factor, mat_bytes,
        processed_detects_cuda + row_offset, processed_valid_cuda + row_offset);

    hipFreeAsync(mat_bytes, kernel_stream);

    row_offset += num_rows;
  }

  hipMemcpyAsync(processed_detects, processed_detects_cuda,
                  sizeof(YoloDetectionCuda) * total_rows,
                  hipMemcpyDeviceToHost, kernel_stream);
  hipMemcpyAsync(processed_valid, processed_valid_cuda,
                  sizeof(bool) * total_rows, hipMemcpyDeviceToHost,
                  kernel_stream);

  hipStreamSynchronize(kernel_stream);

  return 0;
}
}
